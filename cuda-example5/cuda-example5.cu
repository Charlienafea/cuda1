#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <algorithm>
#include <chrono>
/**
 * Matrix multiplication using shared memory.
 * The matrix is assumed to be square.
 */
using Duration = std::chrono::duration<double, std::milli>;

#define TIMEIT(dur,...)\
   {\
    auto start = std::chrono::high_resolution_clock::now();\
    __VA_ARGS__\
    auto end = std::chrono::high_resolution_clock::now();\
     dur = std::chrono::duration<double, std::milli>(end - start);\
}
#define BLOCK_SIZE 32
__global__ void mult(float* da, float* db, float* dc, int width) {

	int by= blockIdx.y;
	int bx = blockIdx.x;
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int row = by * BLOCK_SIZE + ty;
	int col = bx * BLOCK_SIZE + tx;
	__shared__ float sa[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float sb[BLOCK_SIZE][BLOCK_SIZE];
	float res = 0.0;
	int ntiles = width / BLOCK_SIZE;
	for (int b = 0; b < ntiles; ++b) {
		
		/* copy from memory to shared memory */
		sa[ty][tx] = da[row * width + b * BLOCK_SIZE + tx];
		sb[ty][tx] = db[(b * BLOCK_SIZE + ty) * width + col];
		
		__syncthreads();
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			res += sa[ty][k] * sb[k][tx];
		}
		__syncthreads();
	}
	dc[row* width + col] = res;
}


int main() {
	hipEvent_t kernel_start,kernel_end;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);


	float* a, * b, * c;
	float* da, * db, * dc;

	const int matrix_width = 1024;
	const int size = matrix_width * matrix_width;
	a = (float*)malloc(size * sizeof(float));
	b = (float*)malloc(size * sizeof(float));
	c = (float*)malloc(size * sizeof(float));
	for (int i = 0; i < size; ++i) {
		a[i] = 1;
		b[i] = 1;
	}
	hipMalloc(&da, size * sizeof(float));
	hipMalloc(&db, size * sizeof(float));
	hipMalloc(&dc, size * sizeof(float));
	hipMemcpy(da, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, size * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridSize(matrix_width/ BLOCK_SIZE, matrix_width / BLOCK_SIZE);
	mult <<<gridSize, blockSize >> > (da, db, dc, matrix_width);
	float time = 0;
	float gpu_time = 0;
	const int num_trials = 500;
	for (int i = 0; i < num_trials; ++i) {
		hipEventRecord(kernel_start,0);
		mult << <gridSize, blockSize >> > (da, db, dc, matrix_width);
		hipEventRecord(kernel_end,0);
		hipEventSynchronize(kernel_end);
		hipEventElapsedTime(&time, kernel_start, kernel_end);
		gpu_time += time;
	}
	gpu_time /= num_trials;
	std::cout << "GPU  time " << gpu_time << '\n';
	hipMemcpy(c, dc, size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; i++) {
		if (c[i] != matrix_width) {
			std::cout << "error\n";
			break;
		}
		else c[i] = 0;
	}
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	Duration d;
	TIMEIT(d,
		for (int i = 0; i < matrix_width; ++i) {
			for (int j = 0; j < matrix_width; ++j)
				for (int k = 0; k < matrix_width; ++k)
					c[i * matrix_width + j] += a[i * matrix_width+ k] * b[matrix_width * k + j];
		}
	)
		
	std::cout << "CPU time " << d.count() << " milliseconds \n";
	std::cout << "gain = " << d.count() / gpu_time << "\n";
	free(a);
	free(b);
	free(c);

}