#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <iostream>
#include <hip/hip_cooperative_groups.h>

#define NUM_BANKS 4
#define LOG_NUM_BANKS 2

#define CONFLICT_FREE_OFFSET(n) \
   ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

#define BLOCK_SIZE 1024

__global__ void scan1(float* x, float* y) {

	__shared__ float tmp[BLOCK_SIZE];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidx = threadIdx.x;

	int ai = tidx;
	int bi = tidx + (BLOCK_SIZE / 2);

	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	tmp[ai + bankOffsetA] = x[ai];
	tmp[bi + bankOffsetB] = x[bi];


	//tmp[tidx] = x[idx]; tmp[tidx + BLOCK_SIZE / 2] = x[idx + BLOCK_SIZE / 2];
	int offset = 1;
	for (int d = BLOCK_SIZE >> 1; d > 0; d >>= 1) {
		__syncthreads();
		int ai = (2 * tidx + 1) * offset - 1;
		int bi = (2 * tidx + 2) * offset - 1;
		ai += CONFLICT_FREE_OFFSET(ai);
		bi += CONFLICT_FREE_OFFSET(bi);
		if (tidx < d) {
			tmp[bi] += tmp[ai];
		}
		offset *= 2;
	}
	/* down sweep */
	if (tidx == 0)tmp[BLOCK_SIZE - 1 + CONFLICT_FREE_OFFSET(BLOCK_SIZE - 1)] = 0;

	for (int d = 1; d < BLOCK_SIZE; d *= 2) {
		offset >>= 1;
		__syncthreads();
		if (tidx < d) {

			int ai = offset * (2 * tidx + 1) - 1;     int bi = offset * (2 * tidx + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			float t = tmp[ai]; tmp[ai] = tmp[bi]; tmp[bi] += t;
		}
	}

	/* since thread idx is not the same as array index this means
	* threads compute values of different index than the ones they store
	* we have to make sure all of them have finishied the computation before
	* we transfer the values back to DRAM
	*/
	__syncthreads();
	y[ai] = tmp[ai + bankOffsetA]; y[bi] = tmp[bi + bankOffsetB];
}
__global__ void scan2(float* x, float* y) {

	__shared__ float tmp[BLOCK_SIZE];
	int idx = threadIdx.x;
	tmp[idx] = x[idx]; tmp[idx + BLOCK_SIZE / 2] = x[idx + BLOCK_SIZE / 2];
	int offset = 1;
	for (int d = BLOCK_SIZE >> 1; d > 0; d >>= 1) {
		__syncthreads();

		if (idx < d) {
			tmp[(2 * idx + 2) * offset - 1] += tmp[(2 * idx + 1) * offset - 1];
		}
		offset *= 2;
	}
	/* down sweep */
	if (idx == 0)tmp[BLOCK_SIZE - 1] = 0;

	for (int d = 1; d < BLOCK_SIZE; d *= 2) {
		offset >>= 1;
		__syncthreads();
		if (idx < d) {

			int ai = offset * (2 * idx + 1) - 1;     int bi = offset * (2 * idx + 2) - 1;

			float t = tmp[ai]; tmp[ai] = tmp[bi]; tmp[bi] += t;
		}
	}

	/* since thread idx is not the same as array index this means
	* threads compute values of different index than the ones they store
	* we have to make sure all of them have finishied the computation before
	* we transfer the values back to DRAM
	*/
	__syncthreads();
	y[idx] = tmp[idx]; y[idx + BLOCK_SIZE / 2] = tmp[idx + BLOCK_SIZE / 2];
}

int main() {
	const int n = 1 << 10;
	const int blockSize = 1 << 10;
	float* x, * y, * dx, * dy;
	hipMalloc(&dx, n * sizeof(float));
	hipMalloc(&dy, n * sizeof(float));
	x = (float*)malloc(n * sizeof(float));
	y = (float*)malloc(n * sizeof(float));
	memset(y, 0, n * sizeof(float));
	for (int i = 0; i < n; ++i)x[i] = 1;
	hipMemcpy(dx, x, n * sizeof(float), hipMemcpyHostToDevice);

	/*hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	const int num_trials = 1000;
	float total1 = 0, time = 0;
	for (int i = 0; i < num_trials; ++i) {
		hipEventRecord(start, 0);*/
		scan1<< <1, BLOCK_SIZE/2 >> > (dx, dy);
	/*	hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&time, start, end);
		total1 += time;
	}
	hipDeviceSynchronize();
	std::cout << "average duration =" << total1 << "\n";
	float total2 = 0;
	for (int i = 0; i < num_trials; ++i) {
		hipEventRecord(start, 0);
		upsweep << <1, n / 2 >> > (dx, dy, n);
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&time, start, end);
		total2 += time;
	}
	std::cout << "average duration =" << total2 << "\n";*/

	hipMemcpy(y, dy, n * sizeof(float), hipMemcpyDeviceToHost);
	//for (int i = 0; i < n; ++i)
	//	if (y[i] != i * (i + 1) / 2)std::cout << "error";
	//for( int i=n-1;i>n-3;--i)
	std::cout << y[n - 1] + x[n - 1] << "\n";
	std::cout << std::endl;
}