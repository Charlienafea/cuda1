#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
/**
 *  mat_mult()->__global__ void
 * Matrix multiplication without using shared memory
 * @param da
 * @param db
 * @param dc
 * @param width
 * @return 
 */
__global__ void mat_mult(float* da, float* db, float* dc, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float result = 0;
    for (int k = 0; k < width; ++k) 
    {
        result += da[row * width + k] * db[k * width + col];
    }
    dc[row * width + col] = result;
}

void time_kernel(float* da, float* db, float* dc, int width,
                         dim3 blocks_per_grid,dim3 threads_per_block) {
    hipEvent_t kernel_start, kernel_end;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_end);
    /* warmup call*/
    mat_mult <<<blocks_per_grid, threads_per_block >> > (da, db, dc, width);
    float time = 0;
    float total = 0;
   
    for (int i = 0; i < 100; ++i) {
        hipEventRecord(kernel_start);
        mat_mult << <blocks_per_grid, threads_per_block>> > (da, db, dc, width);
        hipEventRecord(kernel_end);
        hipEventSynchronize(kernel_end);
        hipEventElapsedTime(&time, kernel_start, kernel_end);
        total += time;
    }
    /* average time in milliseconds */
    std::cout << "time " << total / 100 << '\n';

}
int main() {
    const int matrix_w = 1024;
    const int msize = matrix_w * matrix_w;
    float* a, * b, * c;

    float* da, * db, * dc;
    a = (float*)malloc(msize * sizeof(float));
    b = (float*)malloc(msize * sizeof(float));
    c = (float*)malloc(msize * sizeof(float));
    for (int i = 0; i < msize; ++i) {
        a[i] = 1;
        b[i] = 1;
    }

    hipMalloc(&da, msize * sizeof(float));
    hipMalloc(&db, msize * sizeof(float));
    hipMalloc(&dc, msize * sizeof(float));
    hipMemcpy(da, a, msize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, msize * sizeof(float), hipMemcpyHostToDevice);

    
    /* total number of threads per block is 1024 which is the maximum */
    dim3 threads_per_block(32, 32);
    dim3 blocks_per_grid(matrix_w / threads_per_block.x, matrix_w/ threads_per_block.y);
    time_kernel(da, db, dc, matrix_w, blocks_per_grid,threads_per_block);
    hipMemcpy(c, dc, msize * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < msize; ++i)
        if (c[i] != 1024)std::cout << "ERROR\n";
    //std::cout << c[i] << ' ';
    std::cout << std::endl;
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);


}