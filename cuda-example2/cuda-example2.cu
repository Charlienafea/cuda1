#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

/**
 *  kernel()->__global__ void
 * add two arrays in parallel
 * @param a 
 * @param b
 * @param c result of addition
 * @return 
 */

__global__ void kernel(float* a, float* b, float* c) {
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main() {
	/* maximum threads per block */
	const int n = 1024;
	float* a, * b, * c;
	float* da, * db, * dc;
	a = (float*)malloc(n * sizeof(float));
	b = (float*)malloc(n * sizeof(float));
	c = (float*)malloc(n * sizeof(float));

	hipMalloc(&da, n * sizeof(float));
	hipMalloc(&db, n * sizeof(float));
	hipMalloc(&dc, n * sizeof(float));
	for (int i = 0; i < n; ++i) {
		a[i] = i;
		b[i] = 2 * i;
	}
	hipMemcpy(da, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, n * sizeof(float), hipMemcpyHostToDevice);

	kernel << <1, n >> > (da, db, dc);
	hipMemcpy(c, dc, n * sizeof(float), hipMemcpyDeviceToHost);
	

	for (int i = 0; i < 10; ++i)
		std::cout << c[i] << ' ';
	std::cout << std::endl;
	free(a);
	free(b);
	free(c);
	hipFree(db);
	hipFree(dc);
	hipFree(da);

}