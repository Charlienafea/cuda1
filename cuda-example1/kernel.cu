#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void kernel(int* a) {
	*a = 17;

}

int main() {
	int a = 3;
	int* da = 0;
	/* allocate memory on device. Note the passing the address of da*/
	hipMalloc(&da, sizeof(int));
	/* launch kernel with 1 block, 1 thread per block */
	kernel << <1, 1 >> > (da);
	/* copy from device to host */
	hipMemcpy(&a, da, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << a << '\n';
	hipFree(da);

}